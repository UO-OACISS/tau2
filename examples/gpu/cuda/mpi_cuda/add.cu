#include "hip/hip_runtime.h"
#include <stdio.h>      // stdio functions are used since C++ streams aren't necessarily thread safe
#include <unistd.h> 
 
// a simple kernel that simply increments each array element by b
__global__ void kernelAddConstant(int *g_a, const int b)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    g_a[idx] += b;
}
 
// a predicate that checks whether each array elemen is set to its index plus b
int correctResult(int *data, const int n, const int b)
{
        for(int i = 0; i < n; i++){
 	//	printf("%d ne %d + %d\n", data[i],i,b);
                if(data[i] != i + b)
                {
			printf("%d ne %d + %d\n", data[i],i,b);
                        return 0;
		}
	}
        return 1;
}

int local_sleep()
{
	sleep(5);
	return 0;
}

 
extern "C" int nv_main(int argc, char *argv[])
{
        int num_gpus = 0;       // number of CUDA GPUs
 
        /////////////////////////////////////////////////////////////////
        // determine the number of CUDA capable GPUs
        //
    hipGetDeviceCount(&num_gpus);
        if(num_gpus < 1)
        {
                printf("no CUDA capable devices were detected\n");
                return 1;
        }
 
        /////////////////////////////////////////////////////////////////
        // display CPU and GPU configuration
        //
    printf("number of CUDA devices:\t%d\n", num_gpus);
    for(int i = 0; i < num_gpus; i++)
    {
        hipDeviceProp_t dprop;
        hipGetDeviceProperties(&dprop, i);
                printf("   %d: %s\n", i, dprop.name);
    }
        printf("---------------------------\n");
 
 
    ////////////////////////////////////////////////////////////////r
    // initialize data
        //
    unsigned int n = num_gpus * 8192;
    unsigned int nbytes = n * sizeof(int);
        int *a = 0;             // pointer to data on the CPU
        int b = 3;              // value by which the array is incremented
        a = (int*)malloc(nbytes);
        if(0 == a)
        {
                printf("couldn't allocate CPU memory\n");
                return 1;
        }
        for(unsigned int i = 0; i < n; i++)
        a[i] = i;
     
 
    ////////////////////////////////////////////////////////////////
        // run as many CPU threads as there are CUDA devices
        //   each CPU thread controls a different device, processing its
        //   portion of the data.  It's possible to use more CPU threads
        //   than there are CUDA devices, in which case several CPU
        //   threads will be allocating resources and launching kernels
        //   on the same device.  For example, try omp_set_num_threads(2*num_gpus);
        //   Recall that all variables declared inside an "omp parallel" scope are
        //   local to each CPU thread
        //
       // omp_set_num_threads(num_gpus);  // create as many CPU threads as there are CUDA devices
    //omp_set_num_threads(2*num_gpus);// create twice as many CPU threads as there are CUDA devices
#pragma omp parallel
    {
        unsigned int cpu_thread_id = 0; // omp_get_thread_num();
                unsigned int num_cpu_threads = 1; // omp_get_num_threads();
// local_sleep();
                // set and check the CUDA device for this CPU thread
                int gpu_id = -1;
                hipSetDevice(cpu_thread_id % num_gpus);        // "% num_gpus" allows more CPU threads than GPU devices
                hipGetDevice(&gpu_id);
 
                printf("CPU thread %d (of %d) uses CUDA device %d\n", cpu_thread_id, num_cpu_threads, gpu_id);
 
                int *d_a = 0;   // pointer to memory on the device associated with this CPU thread
                int *sub_a = a + cpu_thread_id * n / num_cpu_threads;   // pointer to this CPU thread's portion of data
                unsigned int nbytes_per_kernel = nbytes / num_cpu_threads;
                dim3 gpu_threads(128);  // 128 threads per block
                dim3 gpu_blocks(n / (gpu_threads.x * num_cpu_threads));
 
          hipMalloc((void**)&d_a, nbytes_per_kernel);
          hipMemset(d_a, 0, nbytes_per_kernel);
          hipMemcpy(d_a, sub_a, nbytes_per_kernel, hipMemcpyHostToDevice);
        kernelAddConstant<<<gpu_blocks, gpu_threads>>>(d_a, b);
 
          hipMemcpy(sub_a, d_a, nbytes_per_kernel, hipMemcpyDeviceToHost);
          hipFree(d_a);
 
 
    }
        printf("---------------------------\n");
 
        if(hipSuccess != hipGetLastError())
                printf("%s\n", hipGetErrorString(hipGetLastError()));
 
 
        ////////////////////////////////////////////////////////////////
        // check the result
        //
    if(correctResult(a, n, b))
        printf("Test PASSED\n");
    else
        printf("Test FAILED\n");
 
    free(a);    // free CPU memory
 
    hipDeviceReset();
 
    return 0;
}
